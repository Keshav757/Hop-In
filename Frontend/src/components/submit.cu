#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void elementWiseMul3D(float *input1, float *input2, float *output, int dataSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    int idx = z * (gridDim.x * blockDim.x * gridDim.y * blockDim.y) + y * (gridDim.x * blockDim.x) + x;
    if (idx < dataSize) output[idx] = input1[idx] * input2[idx];
}

__global__ void elementWiseMul2D(float *input1, float *input2, float *output, int dataSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * (gridDim.x * blockDim.x) + x;
    if (idx < dataSize) output[idx] = input1[idx] * input2[idx];
}

__global__ void elementWiseMul1D(float *input1, float *input2, float *output, int dataSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dataSize) output[idx] = input1[idx] * input2[idx];
}

void initializeData(float *data, int size) {
    for (int i = 0; i < size; i++) {
        data[i] = rand() % 10;
    }
}

void printData(float *data, int size) {
    for (int i = 0; i < size; i++) {
        printf("%.2f ", data[i]);
    }
    printf("\n");
}

int main() {
    int dataSize = 1024;
    size_t size = dataSize * sizeof(float);
    float *h_input1, *h_input2, *h_output;
    float *d_input1, *d_input2, *d_output;

    h_input1 = (float *)malloc(size);
    h_input2 = (float *)malloc(size);
    h_output = (float *)malloc(size);

    initializeData(h_input1, dataSize);
    initializeData(h_input2, dataSize);

    hipMalloc(&d_input1, size);
    hipMalloc(&d_input2, size);
    hipMalloc(&d_output, size);

    hipMemcpy(d_input1, h_input1, size, hipMemcpyHostToDevice);
    hipMemcpy(d_input2, h_input2, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks(32, 32);
    elementWiseMul2D<<<numBlocks, threadsPerBlock>>>(d_input1, d_input2, d_output, dataSize);
    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);

    printData(h_output, dataSize);

    hipFree(d_input1);
    hipFree(d_input2);
    hipFree(d_output);
    free(h_input1);
    free(h_input2);
    free(h_output);
    return 0;
}


//Question 2

__global__ void rowConvolution(float *input, float *output, float *mask, int n) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < n && y < n) {
        float sum = 0.0;
        for (int i = -1; i <= 1; i++) {
            int idx = x + i;
            if (idx >= 0 && idx < n) sum += mask[i + 1] * input[y * n + idx];
        }
        output[y * n + x] = sum;
    }
}

__global__ void colConvolution(float *input, float *output, float *mask, int n) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < n && y < n) {
        float sum = 0.0;
        for (int i = -1; i <= 1; i++) {
            int idx = y + i;
            if (idx >= 0 && idx < n) sum += mask[i + 1] * input[idx * n + x];
        }
        output[y * n + x] = sum;
    }
}

int main() {
    int n = 3;
    float h_input[9] = {3, 3, 3, 3, 3, 3, 3, 3, 3};
    float h_mask[3] = {0.0, 1.0, 0.0};
    float h_output[9];

    float *d_input, *d_output, *d_mask;
    hipMalloc(&d_input, 9 * sizeof(float));
    hipMalloc(&d_output, 9 * sizeof(float));
    hipMalloc(&d_mask, 3 * sizeof(float));

    hipMemcpy(d_input, h_input, 9 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, h_mask, 3 * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(3, 3);
    dim3 numBlocks(1, 1);
    rowConvolution<<<numBlocks, threadsPerBlock>>>(d_input, d_output, d_mask, n);
    hipMemcpy(h_output, d_output, 9 * sizeof(float), hipMemcpyDeviceToHost);

    printf("Row-wise Convolution:\n");
    printData(h_output, 9);

    colConvolution<<<numBlocks, threadsPerBlock>>>(d_output, d_input, d_mask, n);
    hipMemcpy(h_output, d_input, 9 * sizeof(float), hipMemcpyDeviceToHost);
    printf("Column-wise Convolution:\n");
    printData(h_output, 9);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_mask);
    return 0;
}
